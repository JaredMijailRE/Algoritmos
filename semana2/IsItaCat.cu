#include "hip/hip_runtime.h"
// IsItaCat.cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

// Dispositivo: función que comprueba el patrón meow en una cadena
__device__ bool check_meow(const char* text, int len) {
    if (len < 4) return false;

    int stage = 0;  // 0=m, 1=e, 2=o, 3=w
    int i = 0;

    // debe empezar con al menos un 'm'/'M'
    if (!(text[0]=='m' || text[0]=='M')) return false;

    while (i < len) {
        char c = text[i];
        switch (stage) {
            case 0:
                if (c=='m' || c=='M') {
                    ++i;
                } else {
                    // pasamos a fase 'e'
                    if (c=='e' || c=='E') {
                        stage = 1;
                        ++i;
                    } else {
                        return false;
                    }
                }
                break;
            case 1:
                if (c=='e' || c=='E') {
                    ++i;
                } else {
                    // pasamos a fase 'o'
                    if (c=='o' || c=='O') {
                        stage = 2;
                        ++i;
                    } else {
                        return false;
                    }
                }
                break;
            case 2:
                if (c=='o' || c=='O') {
                    ++i;
                } else {
                    // pasamos a fase 'w'
                    if (c=='w' || c=='W') {
                        stage = 3;
                        ++i;
                    } else {
                        return false;
                    }
                }
                break;
            case 3:
                // solo 'w'/'W' hasta el final
                if (c=='w' || c=='W') {
                    ++i;
                } else {
                    return false;
                }
                break;
        }
    }

    // la última fase debe ser la 3 y haber leído todo
    return (stage == 3);
}

__global__ void isCatKernel(const char* texts, const int* offsets, const int* lengths, bool* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const char* my_text = texts + offsets[idx];
    int len = lengths[idx];
    result[idx] = check_meow(my_text, len);
}

int main(){
    int t;
    if (scanf("%d", &t)!=1) return 0;

    // Host arrays
    std::vector<int> h_len(t), h_off(t);
    std::vector<char> h_text; 
    h_text.reserve(t*50 + 1);

    // Leemos cada caso y concatenamos en un solo buffer
    int offset = 0;
    for(int i = 0; i < t; ++i){
        int n; scanf("%d", &n);
        h_len[i] = n;
        h_off[i] = offset;
        // leemos la cadena (sin espacios)
        std::string s;
        scanf("%s", &s[0]);
        // asegurarnos de que s tenga tamaño n
        s.resize(n);
        // copiamos a buffer
        for(char c: s){
            h_text.push_back(c);
        }
        offset += n;
    }

    // Device arrays
    char*   d_text;
    int*    d_off;
    int*    d_len;
    bool*   d_res;

    hipMalloc(&d_text, h_text.size() * sizeof(char));
    hipMalloc(&d_off,  t     * sizeof(int));
    hipMalloc(&d_len,  t     * sizeof(int));
    hipMalloc(&d_res,  t     * sizeof(bool));

    hipMemcpy(d_text, h_text.data(), h_text.size() * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_off,  h_off.data(),  t     * sizeof(int),  hipMemcpyHostToDevice);
    hipMemcpy(d_len,  h_len.data(),  t     * sizeof(int),  hipMemcpyHostToDevice);

    // Lanzamos un grid de (t) hilos
    int threads = 256;
    int blocks  = (t + threads - 1) / threads;
    isCatKernel<<<blocks, threads>>>(d_text, d_off, d_len, d_res);

    hipDeviceSynchronize();

    // Copiamos resultados de vuelta
    std::vector<bool> h_res(t);
    hipMemcpy(h_res.data(), d_res, t * sizeof(bool), hipMemcpyDeviceToHost);

    // Imprimimos YES/NO
    for(bool ok: h_res){
        printf("%s\n", ok ? "YES" : "NO");
    }

    // Liberamos
    hipFree(d_text);
    hipFree(d_off);
    hipFree(d_len);
    hipFree(d_res);

    return 0;
}
