
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 27  // 3^3
#define ALFABETO "ABC"
#define ALFABETO_LEN 3
#define CADENA_LEN 3

__global__ void generar_cadenas_cuda(char *resultados) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    const char *alfabeto = ALFABETO;
    int id = idx; 

    for (int i = CADENA_LEN - 1; i >= 0; i--) {
        resultados[idx * (CADENA_LEN + 1) + i] = alfabeto[id % ALFABETO_LEN];
        id /= ALFABETO_LEN;
    }

    resultados[idx * (CADENA_LEN + 1) + CADENA_LEN] = '\0';
}


int main() {
    char *d_resultados;
    char h_resultados[N * (CADENA_LEN + 1)];

    // Reservar memoria en GPU
    hipMalloc((void **)&d_resultados, sizeof(char) * N * (CADENA_LEN + 1));

    // Lanzar kernel
    generar_cadenas_cuda<<<1, N>>>(d_resultados);

    // Copiar resultados a la CPU
    hipMemcpy(h_resultados, d_resultados, sizeof(char) * N * (CADENA_LEN + 1), hipMemcpyDeviceToHost);

    // Imprimir resultados
    printf("Cadenas generadas en GPU:\n");
    for (int i = 0; i < N; i++) {
        printf("%s\n", &h_resultados[i * (CADENA_LEN + 1)]);
    }

    // Liberar memoria
    hipFree(d_resultados);
    return 0;
}
