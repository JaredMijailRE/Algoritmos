#include <stdio.h>
#include <hip/hip_runtime.h>

// merge kernel realiza un sort en un area especifica del array
// recibe un puntero al arreglo de entrada, salida, cantidad de elementos y paso
__global__ void mergeKernel(int *d_src, int *d_dst, int n, int step)
{

    // identificador par subarregloas a funcionar
    int pairIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int left = pairIdx * (2 * step);
    if (left >= n)
        return; // si se sale de la longitud del array, nada que hacer

    // ajuste comienzo y fin sub arreglos
    int mid = left + step;
    int right = left + 2 * step;
    if (mid > n)
        mid = n;
    if (right > n)
        right = n;

    int i = left;
    int j = mid;
    int k = left;

    // funcion sub arreglos
    while (i < mid && j < right)
    {
        if (d_src[i] <= d_src[j])
        {
            d_dst[k++] = d_src[i++];
        }
        else
        {
            d_dst[k++] = d_src[j++];
        }
    }

    while (i < mid)
    {
        d_dst[k++] = d_src[i++];
    }
    while (j < right)
    {
        d_dst[k++] = d_src[j++];
    }
}

int main()
{

    int h_arr[] = {1, 5, 3, 7, 8, 9, 5, 6, 9, -10, 834, 83247, -123, -1, 0};
    int n = sizeof(h_arr) / sizeof(h_arr[0]);

    int *d_src = nullptr;
    int *d_dst = nullptr;
    size_t bytes = n * sizeof(int);
    // alocamos memoria en GPU para los dos arreglos que necesitamos
    hipMalloc((void **)&d_src, bytes);
    hipMalloc((void **)&d_dst, bytes);
    // copiamos nuestro array
    hipMemcpy(d_src, h_arr, bytes, hipMemcpyHostToDevice);

    int step = 1;
    bool srcIsInput = true;
    int passCount = 0;

    while (step < n)
    {

        int numPairs = (n + (2 * step - 1)) / (2 * step);
        if (numPairs <= 0)
            break;

        int blockSize = 128;
        int gridSize = (numPairs + blockSize - 1) / blockSize;

        int *d_input = (srcIsInput ? d_src : d_dst);
        int *d_output = (srcIsInput ? d_dst : d_src);

        // se llama a nuestro kernel
        mergeKernel<<<gridSize, blockSize>>>(d_input, d_output, n, step);
        hipGetLastError();

        srcIsInput = !srcIsInput;
        step <<= 1;
        passCount++;
    }

    int *d_result = ((passCount & 1) == 0 ? d_src : d_dst);

    hipMemcpy(h_arr, d_result, bytes, hipMemcpyDeviceToHost);
    printf("Sorted array:\n");

    // bucle para imprimir el arreglo
    for (int i = 0; i < n; i++)
    {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    // liberamos memoria GPU
    hipFree(d_src);
    hipFree(d_dst);
    return 0;
}
